﻿#include "hip/hip_runtime.h"
#include "defns.h"
#include "gmfft.h"
#include <chrono>
#include <iostream>
#include <stdio.h>


int srwlUtiFFT(char *pcData, char typeData, double *arMesh, int nMesh,
               int dir) {
  int locErNo = 0;
  try {
    long nx = (long)arMesh[2];
    long ny = 1;
    if (nMesh >= 6)
      ny = (long)arMesh[5];

    int dimFFT = 1;
    if (ny > 1)
      dimFFT = 2;
    // float *pfData = (float*)pcData; //OC31012019 (commented-out)

    if (dimFFT == 1) {
      CGenMathFFT1DInfo FFT1DInfo;

      if (typeData == 'f') {
        FFT1DInfo.pInData = (float *)pcData;
        FFT1DInfo.pOutData = FFT1DInfo.pInData;
      }
#ifdef _FFTW3 // OC31012019
      else if (typeData == 'd') {
        FFT1DInfo.pdInData = (double *)pcData;
        FFT1DInfo.pdOutData = FFT1DInfo.pdInData;
      }
#endif

      FFT1DInfo.Dir = (char)dir;
      FFT1DInfo.xStart = arMesh[0];
      FFT1DInfo.xStep = arMesh[1];
      FFT1DInfo.Nx = nx;
      FFT1DInfo.HowMany = 1;
      FFT1DInfo.UseGivenStartTrValue = 0;

      CGenMathFFT1D FFT1D;
      if (locErNo = FFT1D.Make1DFFT(FFT1DInfo))
        return locErNo;

      arMesh[0] = FFT1DInfo.xStartTr;
      arMesh[1] = FFT1DInfo.xStepTr;
    } else {
      CGenMathFFT2DInfo FFT2DInfo;
      // FFT2DInfo.pData = pfData;
      if (typeData == 'f') // OC31012019
      {
        FFT2DInfo.pData = (float *)pcData;
      }
#ifdef _FFTW3 // OC31012019
      else if (typeData == 'd') {
        FFT2DInfo.pdData = (double *)pcData;
      }
#endif

      FFT2DInfo.Dir = (char)dir;
      FFT2DInfo.xStart = arMesh[0];
      FFT2DInfo.xStep = arMesh[1];
      FFT2DInfo.Nx = nx;
      FFT2DInfo.yStart = arMesh[3];
      FFT2DInfo.yStep = arMesh[4];
      FFT2DInfo.Ny = ny;
      FFT2DInfo.UseGivenStartTrValues = 0;

      CGenMathFFT2D FFT2D;
      if (locErNo = FFT2D.Make2DFFT(FFT2DInfo))
        return locErNo;

      arMesh[0] = FFT2DInfo.xStartTr;
      arMesh[1] = FFT2DInfo.xStepTr;
      arMesh[3] = FFT2DInfo.yStartTr;
      arMesh[4] = FFT2DInfo.yStepTr;
    }
  } catch (int erNo) {
    return erNo;
  }
  return 0;
}

int main() {
  double xStart = -5;
  double xRange = 10;
  long xNp = 100000000;
  double xStep = xRange / (xNp - 1);
  double mesh[3] = {xStart, xStep, xNp};
  float *input_data;
  float *input_data_cpy;
  int runs = 50;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMallocManaged(&input_data, 2 * xNp * sizeof(float));
  hipMallocManaged(&input_data_cpy, 2 * xNp * sizeof(float));
  memset(input_data, 0, sizeof(float) * 2 * xNp);

  int x0 = (int)((-0.5 - xStart) / xStep);
  int x1 = (int)((0.5 - xStart) / xStep);
  for (unsigned int i = x0; i < x1; i++) {
    input_data[2 * i] = 1;
  }

  double net_time = 0;
  double net_time_gpu = 0;

  hipMemcpy(input_data_cpy, input_data, 2 * xNp * sizeof(float),
             hipMemcpyDefault);
  srwlUtiFFT(reinterpret_cast<char *>(input_data), 'f', mesh, 3,
             1); // warm up run

  std::cout << "Starting benchmark " << std::endl;

  for (int i = 0; i < runs; i++) {
    hipMemcpy(input_data, input_data_cpy, 2 * xNp * sizeof(float),
               hipMemcpyDefault);
    
    hipEventRecord(start);

    std::chrono::high_resolution_clock::time_point t0 = std::chrono::high_resolution_clock::now();

    srwlUtiFFT(reinterpret_cast<char *>(input_data), 'f', mesh, 3, 1);
    
    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> ts = std::chrono::duration_cast<std::chrono::duration<double>>(t1 - t0);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Measure gpu time
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    net_time_gpu += ms;

    net_time += ts.count();
  }

  net_time_gpu /= runs;
  net_time /= runs;
  net_time *= 1000;

  std::cout << "Benchmark took " << net_time << " ms as measured from CPU." << std::endl;
  std::cout << "Benchmark took " << net_time_gpu << " ms as measured from GPU." << std::endl;

  // hipDeviceReset must be called before exiting in order for profiling and
  // tracing tools such as Nsight and Visual Profiler to show complete traces.
  hipError_t cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!");
    return 1;
  }

  return 0;
}
